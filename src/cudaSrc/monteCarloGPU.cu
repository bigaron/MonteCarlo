#include "hip/hip_runtime.h"
#include "glew.h"
#include "GLFW/glfw3.h"
#include "glm/glm.hpp"

#include <hip/hip_runtime.h>
#include <cudaSrc/hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

// For threadIdx, blockDim, blockIdx
#include <>


__global__ void monteCarloKernel(float4* h_odata, int imgw) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x, iy = blockIdx.y * blockDim.y + threadIdx.y;
	float val = ix / (float)imgw;
	h_odata[iy * imgw + ix] = make_float4(val, 1.0f - val, 1.0f - val, 1.0);
}

extern "C" void computeMonteCarlo(float4* h_odata, dim3 grid, dim3 block, int imgW) {
	monteCarloKernel <<< grid, block >> > (h_odata, imgW);
}